// #include <stdio.h>
// #include <stdlib.h>
// #include <math.h>
// #include <time.h>
// #define SPHERE_NUM 20
// #define rnd( x ) (x * rand() / RAND_MAX)
// #define INF 2e10f
// #define DIM 2048
// #define N (2048*2048)
// #define THREADS_PER_BLOCK 512
// typedef struct Sphere {
// 	float	r,g,b;
//     float   radius;
//     float   x,y,z;
//     __device__ float hit( float ox, float oy, float *n ) {
//         float dx = ox - x;
//         float dy = oy - y;
//         if (dx*dx + dy*dy < radius*radius) {
//             float dz = sqrtf( radius*radius - dx*dx - dy*dy );
//             *n = dz / sqrtf( radius * radius );
//             return dz + z;
//         }
//         return -INF;
//     }
// } Sphere;
// __global__ void cuda_ray(Sphere* s, unsigned char* ptr) {
// 	int ix = threadIdx.x + blockIdx.x * blockDim.x;
// 	int iy = threadIdx.y + blockIdx.y * blockDim.y;

// 	float ox = (ix - DIM / 2), oy = (iy - DIM / 2), maxz = -INF;
// 	float rgb[4] = {0, 0, 0, 1};

// 	for (int k = 0; k < SPHERE_NUM; k++) {
// 		float n, t = s[k].hit(ox, oy, &n);
// 		if (t > maxz) {
// 			float fscale = n;
// 			rgb[0] = s[k].r * fscale;
// 			rgb[1] = s[k].g * fscale;
// 			rgb[2] = s[k].b * fscale;
// 			maxz = t;
// 		}
// 	}

// 	int offset = ix + iy * DIM;
// 	for (int k = 0; k < 4; k++)	{
// 		ptr[offset * 4 + k] = (int)(rgb[k] * 255);
// 	}
// }
// void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp) {
// 	int i,x,y;
// 	fprintf(fp,"P3\n");
// 	fprintf(fp,"%d %d\n",xdim, ydim);
// 	fprintf(fp,"255\n");
// 	for (y = 0; y < ydim; y++) {
// 		for (x = 0; x < xdim; x++) {
// 			i = x + y * xdim;
// 			fprintf(fp,"%d %d %d ", bitmap[4*i], bitmap[4*i + 1], bitmap[4*i + 2]);
// 		}
// 		fprintf(fp,"\n");
// 	}
// }
// int main(void) {
//     int sph_size = 4 * sizeof(Sphere) * SPHERE_NUM, bmp_size = 4 * sizeof(unsigned char) * DIM * DIM * 4;
// 	unsigned char* bitmap = (unsigned char *) malloc(bmp_size), *d_bitmap;
//     FILE* rp = fopen("result_cuda.ppm", "w");
// 	Sphere *spheres = (Sphere *)malloc(sph_size), *d_spheres;
// 	srand((unsigned int)time(NULL));
// 	for (int i = 0; i < SPHERE_NUM; i++) {
// 		spheres[i].r = rnd( 1.0f );
// 		spheres[i].g = rnd( 1.0f );
// 		spheres[i].b = rnd( 1.0f );
// 		spheres[i].x = rnd( 2000.0f ) - 1000;
// 		spheres[i].y = rnd( 2000.0f ) - 1000;
// 		spheres[i].z = rnd( 2000.0f ) - 1000;
// 		spheres[i].radius = rnd( 200.0f ) + 40;
// 	}
	
//     cudaMalloc((void **)&d_spheres, sph_size);
//     cudaMalloc((void **)&d_bitmap, bmp_size);

//     cudaMemcpy(d_spheres, spheres, sph_size, cudaMemcpyHostToDevice);
//     cudaMemcpy(d_bitmap, bitmap, bmp_size, cudaMemcpyHostToDevice);

// 	cudaEvent_t startTime, endTime;
// 	cudaEventCreate(&startTime);
// 	cudaEventCreate(&endTime);

// 	cudaEventRecord(startTime);
	
// 	dim3 blocks(1 << 6, 1 << 6, 1);
// 	dim3 threads(1 << 5, 1 << 5, 1);
// 	cuda_ray<<<blocks, threads>>>(d_spheres, d_bitmap);

//     cudaEventRecord(endTime);

// 	cudaEventSynchronize(endTime);
// 	float excTime = 0.0;
// 	cudaEventElapsedTime(&excTime, startTime, endTime);

//     cudaMemcpy(bitmap, d_bitmap, bmp_size, cudaMemcpyDeviceToHost);
//     ppm_write(bitmap, DIM, DIM, rp);

// 	cudaEventDestroy(startTime);
// 	cudaEventDestroy(endTime);
// 	fclose(rp);
// 	free(bitmap); free(spheres);
//     cudaFree(d_bitmap); cudaFree(d_spheres);
//     printf("%lfsec\n",  (double)excTime / 1000.0);
//     return 0;
// }


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#pragma warning(disable: 4819)
#define SPHERE_NUM 20
#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048
typedef struct Sphere {
	float	rgb[3];
    float   radius;
    float   x,y,z;
    __device__ float hit( float ox, float oy, float *n ) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
} Sphere;
__global__ void cuda_ray(Sphere* s, unsigned char* ptr) {
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	int iz = threadIdx.z + blockIdx.z * blockDim.z;
	
	float ox = (ix - DIM / 2), oy = (iy - DIM / 2), maxz = -INF;
	float rgb = iz != 3 ? 0 : 1;

	for (int k = 0; k < SPHERE_NUM; k++) {
		float n, t = s[k].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			rgb = s[k].rgb[iz] * fscale;
			maxz = t;
		}
	}

	ptr[(ix + iy * DIM) * 4 + iz] = (int)(rgb * 255);
}
void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp) {
	int i,x,y;
	fprintf(fp,"P3\n");
	fprintf(fp,"%d %d\n",xdim, ydim);
	fprintf(fp,"255\n");
	for (y = 0; y < ydim; y++) {
		for (x = 0; x < xdim; x++) {
			i = x + y * xdim;
			fprintf(fp,"%d %d %d ", bitmap[4*i], bitmap[4*i + 1], bitmap[4*i + 2]);
		}
		fprintf(fp,"\n");
	}
}
int main(void) {
    unsigned int sph_size = 4 * sizeof(Sphere) * SPHERE_NUM, bmp_size = 4 * sizeof(unsigned char) * DIM * DIM * 4;
	unsigned char* bitmap = (unsigned char *) malloc(bmp_size), *d_bitmap;
	char file_name[20] = "result_cuda.ppm";
    FILE* rp = fopen(file_name, "w");
	Sphere *spheres = (Sphere *)malloc(sph_size), *d_spheres;
	srand((unsigned int)time(NULL));
	for (int i = 0; i < SPHERE_NUM; i++) {
		for(int j = 0; j < 3; j++) {
			spheres[i].rgb[j] = rnd(1.0f);
		}
		spheres[i].x = rnd( 2000.0f ) - 1000;
		spheres[i].y = rnd( 2000.0f ) - 1000;
		spheres[i].z = rnd( 2000.0f ) - 1000;
		spheres[i].radius = rnd( 200.0f ) + 40;
	}
	
    hipMalloc((void **)&d_spheres, sph_size);
    hipMalloc((void **)&d_bitmap, bmp_size);

    hipMemcpy(d_spheres, spheres, sph_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bitmap, bitmap, bmp_size, hipMemcpyHostToDevice);

	hipEvent_t startTime, endTime;
	hipEventCreate(&startTime);
	hipEventCreate(&endTime);

	hipEventRecord(startTime);
	
	dim3 blocks(1 << 7, 1 << 7, 2);
	dim3 threads(1 << 4, 1 << 4, 2);
	cuda_ray<<<blocks, threads>>>(d_spheres, d_bitmap);
	
	hipEventRecord(endTime);

	hipEventSynchronize(endTime);
	
	float excTime = 0.0;
	hipEventElapsedTime(&excTime, startTime, endTime);

    hipMemcpy(bitmap, d_bitmap, bmp_size, hipMemcpyDeviceToHost);
    ppm_write(bitmap, DIM, DIM, rp);

	hipEventDestroy(startTime);
	hipEventDestroy(endTime);
	fclose(rp);
	free(bitmap); free(spheres);
    hipFree(d_bitmap); hipFree(d_spheres);
    printf("CUDA ray tracing: %lfsec\n",  (double)excTime / 1000.0);
	printf("[%s] generated\n", file_name);
    return 0;
}