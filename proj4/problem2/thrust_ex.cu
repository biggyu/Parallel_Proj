#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/sequence.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
template <typename T>
struct square {

    double num_step = 1 << 30;

    __host__ __device__
    double operator()(const T& x) const {
        // printf("%f\n", 4 / pow(x, 2));
        double sum = 0;
        for(int i = 0; i < 4; i++ ) {
            double step = 1.0 / num_step;
            sum += 4.0 / (1 + pow(((x + i * (1 << 28)) + 0.5) * step, 2));
        }
        return sum;
    }
};
int main () {
    long num_step = 1 << 28;
    thrust::device_vector<long> X(num_step);

    square<double>         unary_op;
    thrust::plus<double>   binary_op;

    thrust::sequence(X.begin(), X.end());
    clock_t start_time = clock();
    double result = thrust::transform_reduce(X.begin(), X.end(), unary_op, (double) 0, binary_op);
    double exc_time = (double) clock() - start_time;
    printf("%lf\n", result / (double) (num_step << 2));
    printf("execution time: %lf sec\n", exc_time / 1000.0);

    return 0;
}