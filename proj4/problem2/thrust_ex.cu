#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/sequence.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
template <typename T>
struct square {

    int num_step = 1 << 30;

    __host__ __device__
    double operator()(const T& x) const {
        double step = 1.0 / num_step;
        return 4.0 / (1 + pow((x + 0.5) * step, 2));
    }
};
int main () {
    int num_step = 1 << 30;
    thrust::device_vector<int> X(num_step);

    square<double>         unary_op;
    thrust::plus<double>   binary_op;

    thrust::sequence(X.begin(), X.end());
    clock_t start_time = clock();
    double result = thrust::transform_reduce(X.begin(), X.end(), unary_op, (double) 0, binary_op);
    double exc_time = (double) (clock() - start_time);
    printf("%lf\n", result / (double) num_step);
    printf("execution time: %lf sec\n", exc_time / 1000.0);

    return 0;
}